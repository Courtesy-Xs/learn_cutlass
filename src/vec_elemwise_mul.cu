#include "hip/hip_runtime.h"
// error: identifier "CUTE_STL_NAMESPACE" is undefined, I suspect it’s a cutlass issue
// #include "cute/numeric/int.hpp"

#include <cute/tensor.hpp>
#include <hip/hip_fp16.h>
#include <cstdlib>
#include <cmath>


using namespace cute;

// Vector Add
// z = ax + by + c
template <int kNumElemPerThread = 8>
__global__ void vector_add_local_tile_multi_elem_per_thread_half(
    half *z, int num, const half *x, const half *y, const half a, const half b, const half c) {
  using namespace cute;

  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx >= num / kNumElemPerThread) { // 未处理非对齐问题
    return;
  }

  Tensor tz = make_tensor(make_gmem_ptr(z), make_shape(num));
  Tensor tx = make_tensor(make_gmem_ptr(x), make_shape(num));
  Tensor ty = make_tensor(make_gmem_ptr(y), make_shape(num));

  Tensor tzr = local_tile(tz, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));
  Tensor txr = local_tile(tx, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));
  Tensor tyr = local_tile(ty, make_shape(Int<kNumElemPerThread>{}), make_coord(idx));

  Tensor txR = make_tensor_like(txr);
  Tensor tyR = make_tensor_like(tyr);
  Tensor tzR = make_tensor_like(tzr);

  // LDG.128
  copy(txr, txR);
  copy(tyr, tyR);

  half2 a2 = {a, a};
  half2 b2 = {b, b};
  half2 c2 = {c, c};

  auto tzR2 = recast<half2>(tzR);
  auto txR2 = recast<half2>(txR);
  auto tyR2 = recast<half2>(tyR);

#pragma unroll
  for (int i = 0; i < size(tzR2); ++i) {
    // two hfma2 instruction
    tzR2(i) = txR2(i) * a2 + (tyR2(i) * b2 + c2);
  }

  auto tzRx = recast<half>(tzR2);

  // STG.128
  copy(tzRx, tzr);
}

int main()
{
  const int kNumel = 1024;
  half *hx, *hy, *hz, *dx, *dy, *dz;

  half a = 1.0;
  half b = 1.0;
  half c = 1.0;

  hipMalloc((void**)(&dx), sizeof(half) * kNumel);
  hipMalloc((void**)(&dy), sizeof(half) * kNumel);
  hipMalloc((void**)(&dz), sizeof(half) * kNumel);

  hipMemset(dx, 0, sizeof(half) * kNumel);
  hipMemset(dy, 0, sizeof(half) * kNumel);
  hipMemset(dz, 0, sizeof(half) * kNumel);

  hx = (half*)(malloc(kNumel * sizeof(half)));
  hy = (half*)(malloc(kNumel * sizeof(half)));  
  hz = (half*)(malloc(kNumel * sizeof(half)));

  std::for_each(hx, hx + kNumel, [](half& ele){ ele = 1.0; });
  std::for_each(hy, hy + kNumel, [](half& ele){ ele = 1.0; });

  hipMemcpy(dx, hx, sizeof(half) * kNumel, hipMemcpyHostToDevice);
  hipMemcpy(dy, hy, sizeof(half) * kNumel, hipMemcpyHostToDevice);

  dim3 grid(1);
  dim3 block(256);

  vector_add_local_tile_multi_elem_per_thread_half<4><<<grid, block, 0, 0>>>(dz, kNumel, dx, dy, a, b, c);

  hipDeviceSynchronize();

  hipMemcpy(hz, dz, sizeof(half) * kNumel, hipMemcpyDeviceToHost);

  // verify
  std::for_each(hz, hz + kNumel, [](half val){ if(std::fabs(static_cast<float>(val)) - 3.0f > 1e-7) { std::cout << static_cast<float>(val) << "\t";} });

  // output
  // std::for_each(hz, hz + kNumel, [](half val){ std::cout << static_cast<float>(val) << "\t";});

  return 0;
}